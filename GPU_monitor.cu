#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <nvml.h>
#include <iostream>
#include <sys/time.h>
#include <unistd.h>

using namespace std;

int stop = 0;
nvmlReturn_t mlResult;
nvmlDevice_t *device;
nvmlMemory_t *memory;
nvmlUtilization_t *utilization;
nvmlTemperatureSensors_t sensorType;
char **name, path[256], save_part[128], saveFile[256];
unsigned int *power, *temperature;
int devs, N_run = 1, r_count = 1;
unsigned int nvmlDevs;


void getDate();
void getTime();
void startup(int argc, char **argv);

void monitora()
{
   struct timeval *tvnow;
   tvnow = (struct timeval*)malloc(sizeof(struct timeval)*nvmlDevs);
    
   for(int i = 0; i < nvmlDevs; i++)
   {
      gettimeofday(&tvnow[i], NULL);
      mlResult = nvmlDeviceGetName(device[i], name[i], 50);
      mlResult = nvmlDeviceGetMemoryInfo(device[i], &memory[i]);

      mlResult =  nvmlDeviceGetUtilizationRates(device[i], &utilization[i]);
      if(NVML_SUCCESS != mlResult)
      {
         printf("Failed to get utilization rates: %s\n", nvmlErrorString(mlResult));
      }
   }
   for(int i = 0; i < nvmlDevs; i++)
   {
      struct tm* tm = localtime(&tvnow[i].tv_sec);
      printf("%3d:%02d:%02d.%06ld  %s  %5d %9.2Lf %9.2Lf %10.2Lf %9i %9i \n", tm->tm_hour, tm->tm_min, tm->tm_sec, tvnow[i].tv_usec, name[i], i,(long double)memory[i].free/1048576.0,(long double)memory[i].used/1048576.0, (long double)memory[i].total/1048576.0, utilization[i].memory, utilization[i].gpu);

   }
}


void getTime()
{
   struct timeval tvnow;
   gettimeofday(&tvnow, NULL);
   struct tm* tm = localtime(&tvnow.tv_sec);

   printf("%d:%02d:%02d.%06ld\n", tm->tm_hour, tm->tm_min, tm->tm_sec, tvnow.tv_usec);

}

int main(int argc, char **argv)
{
   mlResult = nvmlInit();
   if(NVML_SUCCESS != mlResult)
      printf("Failed to Initialize NVML: %s\n", nvmlErrorString(mlResult));

   nvmlDeviceGetCount(&nvmlDevs);
   device = (nvmlDevice_t *)malloc(sizeof(nvmlDevice_t)*nvmlDevs);
   memory = (nvmlMemory_t*)malloc(sizeof(nvmlMemory_t)*nvmlDevs);
   name = (char**)malloc(sizeof(char*)*nvmlDevs);
   utilization = (nvmlUtilization_t*)malloc(sizeof(nvmlUtilization_t)*nvmlDevs);

   for(int i = 0; i < nvmlDevs; i++)
   {
      mlResult = nvmlDeviceGetHandleByIndex(i, &device[i]);
      name[i] = (char*)malloc(sizeof(char)*50);
   }
   monitora(); 

   mlResult = nvmlShutdown();
   if(NVML_SUCCESS != mlResult)
   {
      printf("Failed to shutdown NVML: %s\n", nvmlErrorString(mlResult));

      printf("Press ENTER to continue...\n");
      getchar();
   }


   return 0;
}
